#include <iostream>
#include <ctime>
#include <chrono>

#include "kernel/matrix.cu"

using namespace std;

void printMatrix(const float* matrix, const int rows, const int columns) {

    for (int i=0; i<rows * columns; i++) {
        if (i % columns == 0) {
            cout << "\n";
        }
        cout << matrix[i] << " ";
    }

    cout << "\n";
}

float* initRandomMatrix(const int rows, const int columns) {
    srand(chrono::high_resolution_clock::now().time_since_epoch().count());

    const int numberOfElements = rows * columns;
    float* matrix = (float*) malloc(numberOfElements * sizeof(float));

    for (int i=0; i<numberOfElements; i++) {
        matrix[i] = static_cast<int>( rand() ) % 10; // TODO back to floats in future
    }

    return matrix;
}

int main(int argc, char* argv[]) {

    const int TILE_SIZE = 16;
    const int size = 4;

    int rowsA = size;
    int colsA = size;
    int rowsB = size;
    int colsB = size;
    int rowsC = rowsA;
    int colsC = colsB;

    float* matrixA = initRandomMatrix(rowsA, colsA);
    float* matrixB = initRandomMatrix(rowsB, colsB);
    float* matrixC = (float*) malloc(rowsC * colsC * sizeof(float));

    printMatrix(matrixA, rowsA, colsC);
    printMatrix(matrixB, rowsB, colsB);

    float* dev_matrixA;
    float* dev_matrixB;
    float* dev_matrixC;

    hipMalloc((void**) &dev_matrixA, rowsA * colsA * sizeof(float));
    hipMalloc((void**) &dev_matrixB, rowsB * colsB * sizeof(float));
    hipMalloc((void**) &dev_matrixC, rowsC * colsC * sizeof(float));

    hipMemcpy(dev_matrixA, matrixA, rowsA * colsA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_matrixB, matrixB, rowsB * colsB * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventRecord(start, 0);

    dim3 gridSize((colsC-1) / TILE_SIZE + 1, (rowsC-1) / TILE_SIZE + 1, 1);
    dim3 blockSize(TILE_SIZE, TILE_SIZE, 1);

    matrixMultiplication<<<gridSize, blockSize>>>(
        dev_matrixC, dev_matrixA, dev_matrixB,
        rowsC, colsC, rowsA, colsA, rowsB, colsB
    );

    hipDeviceSynchronize();

    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(matrixC, dev_matrixC, rowsC * colsC * sizeof(float), hipMemcpyDeviceToHost);

    cout << "\nResult:";
    printMatrix(matrixC, rowsC, colsC);

    cout << "Elapsed time: " << elapsedTime << " ms\n";

    hipFree(dev_matrixA);
    hipFree(dev_matrixB);
    hipFree(dev_matrixC);

    free(matrixA);
    free(matrixB);
    free(matrixC);

    return EXIT_SUCCESS;
}